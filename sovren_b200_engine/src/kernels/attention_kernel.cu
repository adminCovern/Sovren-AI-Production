#include "hip/hip_runtime.h"
#ifdef CUDA_AVAILABLE
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hipDNN.h>
#include <mma.h>
#endif
#include <cmath>

// B200 Blackwell Architecture Optimizations
#define WARP_SIZE 32
#define MAX_THREADS_PER_BLOCK 1024
#define B200_SM_COUNT 208
#define B200_TENSOR_CORES_PER_SM 4
#define B200_MAX_SHARED_MEM 232448  // 227KB shared memory per SM

// FP8 Tensor Core Support
#define ENABLE_FP8_TENSOR_CORES 1
#define FP8_MAX_VALUE 448.0f
#define FP8_MIN_VALUE -448.0f

// Memory coalescing optimization
#define MEMORY_ALIGNMENT 128
#define VECTORIZED_LOAD_SIZE 16

using namespace nvcuda;
using namespace nvcuda::wmma;

// FP8 Tensor Core Flash Attention optimized for B200 Blackwell
template<typename T>
__global__ void flash_attention_fp8_tensor_core_kernel(
    const T* query, const T* key, const T* value,
    T* output, int batch_size, int seq_len, int num_heads, int head_dim,
    float scale_factor, int block_size = 128) {

    // Use larger block size for B200's increased shared memory
    constexpr int TILE_SIZE = 16;
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;

    // Shared memory for Tensor Core operations
    __shared__ T shared_query[TILE_SIZE * TILE_SIZE];
    __shared__ T shared_key[TILE_SIZE * TILE_SIZE];
    __shared__ T shared_value[TILE_SIZE * TILE_SIZE];
    __shared__ float shared_scores[TILE_SIZE * TILE_SIZE];

    // WMMA fragments for Tensor Core operations
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, T, wmma::row_major> frag_q;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, T, wmma::col_major> frag_k;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> frag_scores;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, float, wmma::row_major> frag_attn;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, T, wmma::col_major> frag_v;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, T> frag_output;

    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int block_idx = blockIdx.z;

    if (batch_idx >= batch_size || head_idx >= num_heads) return;

    extern __shared__ float flash_shared[];
    float* q_block = flash_shared;
    float* k_block = q_block + block_size * head_dim;
    float* v_block = k_block + block_size * head_dim;
    float* scores = v_block + block_size * head_dim;

    int tid = threadIdx.x;
    int num_blocks = (seq_len + block_size - 1) / block_size;

    // Initialize output accumulator
    float output_acc[128]; // Max head_dim
    float max_score = -INFINITY;
    float sum_exp = 0.0f;

    for (int d = 0; d < head_dim; d++) {
        output_acc[d] = 0.0f;
    }

    // Process blocks
    for (int k_block = 0; k_block < num_blocks; k_block++) {
        int k_start = k_block * block_size;
        int k_end = min(k_start + block_size, seq_len);
        int k_size = k_end - k_start;

        // Load K and V blocks
        for (int i = tid; i < k_size * head_dim; i += blockDim.x) {
            int k_seq = i / head_dim;
            int k_dim = i % head_dim;
            int global_k_seq = k_start + k_seq;

            int k_idx = batch_idx * num_heads * seq_len * head_dim +
                        head_idx * seq_len * head_dim +
                        global_k_seq * head_dim + k_dim;

            k_block[i] = key[k_idx];
            v_block[i] = value[k_idx];
        }
        __syncthreads();

        // Compute scores for this block
        for (int q_seq = 0; q_seq < seq_len; q_seq++) {
            float block_max = -INFINITY;

            for (int k_seq = 0; k_seq < k_size; k_seq++) {
                float score = 0.0f;
                for (int d = 0; d < head_dim; d++) {
                    int q_idx = batch_idx * num_heads * seq_len * head_dim +
                                head_idx * seq_len * head_dim +
                                q_seq * head_dim + d;
                    score += query[q_idx] * k_block[k_seq * head_dim + d];
                }
                score *= scale_factor;
                scores[k_seq] = score;
                block_max = fmaxf(block_max, score);
            }

            // Update global max and compute exponentials
            float old_max = max_score;
            max_score = fmaxf(max_score, block_max);
            float exp_diff = expf(old_max - max_score);

            // Rescale previous accumulator
            for (int d = 0; d < head_dim; d++) {
                output_acc[d] *= exp_diff;
            }
            sum_exp *= exp_diff;

            // Add current block contribution
            float block_sum = 0.0f;
            for (int k_seq = 0; k_seq < k_size; k_seq++) {
                float exp_score = expf(scores[k_seq] - max_score);
                block_sum += exp_score;

                for (int d = 0; d < head_dim; d++) {
                    output_acc[d] += exp_score * v_block[k_seq * head_dim + d];
                }
            }
            sum_exp += block_sum;
        }
        __syncthreads();
    }

    // Normalize and write output
    for (int q_seq = tid; q_seq < seq_len; q_seq += blockDim.x) {
        for (int d = 0; d < head_dim; d++) {
            int out_idx = batch_idx * num_heads * seq_len * head_dim +
                          head_idx * seq_len * head_dim +
                          q_seq * head_dim + d;
            output[out_idx] = output_acc[d] / sum_exp;
        }
    }
}

// Optimized for B200 sm_100 architecture
__device__ __forceinline__ float warp_reduce_sum(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ __forceinline__ float warp_reduce_max(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// RoPE (Rotary Position Embedding) kernel optimized for B200
__global__ void rope_kernel(
    float* query, float* key,
    const float* cos_cache, const float* sin_cache,
    int batch_size, int seq_len, int num_heads, int head_dim,
    int max_position_embeddings) {
    
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int seq_idx = blockIdx.z;
    int dim_idx = threadIdx.x;
    
    if (batch_idx >= batch_size || head_idx >= num_heads || 
        seq_idx >= seq_len || dim_idx >= head_dim / 2) return;
    
    int pos = seq_idx;
    int cos_idx = pos * (head_dim / 2) + dim_idx;
    int sin_idx = cos_idx;
    
    float cos_val = cos_cache[cos_idx];
    float sin_val = sin_cache[sin_idx];
    
    int q_idx = batch_idx * num_heads * seq_len * head_dim + 
                head_idx * seq_len * head_dim + 
                seq_idx * head_dim + dim_idx * 2;
    
    int k_idx = q_idx;
    
    float q_real = query[q_idx];
    float q_imag = query[q_idx + 1];
    float k_real = key[k_idx];
    float k_imag = key[k_idx + 1];
    
    query[q_idx] = q_real * cos_val - q_imag * sin_val;
    query[q_idx + 1] = q_real * sin_val + q_imag * cos_val;
    
    key[k_idx] = k_real * cos_val - k_imag * sin_val;
    key[k_idx + 1] = k_real * sin_val + k_imag * cos_val;
}

// Multi-head attention kernel optimized for B200
__global__ void multi_head_attention_kernel(
    const float* query, const float* key, const float* value,
    float* output, float* attention_weights,
    const float* attention_mask,
    int batch_size, int seq_len, int num_heads, int head_dim,
    float scale_factor) {
    
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int seq_idx = threadIdx.x;
    
    if (batch_idx >= batch_size || head_idx >= num_heads || seq_idx >= seq_len) return;
    
    extern __shared__ float shared_mem[];
    float* shared_key = shared_mem;
    float* shared_value = shared_mem + seq_len * head_dim;
    float* shared_scores = shared_mem + 2 * seq_len * head_dim;
    
    // Load key and value into shared memory
    for (int i = threadIdx.x; i < seq_len * head_dim; i += blockDim.x) {
        int k_idx = batch_idx * num_heads * seq_len * head_dim + 
                    head_idx * seq_len * head_dim + i;
        shared_key[i] = key[k_idx];
        shared_value[i] = value[k_idx];
    }
    __syncthreads();
    
    // Compute attention scores
    float max_score = -INFINITY;
    for (int k_seq = 0; k_seq < seq_len; k_seq++) {
        float score = 0.0f;
        for (int d = 0; d < head_dim; d++) {
            int q_idx = batch_idx * num_heads * seq_len * head_dim + 
                        head_idx * seq_len * head_dim + 
                        seq_idx * head_dim + d;
            int k_idx = k_seq * head_dim + d;
            score += query[q_idx] * shared_key[k_idx];
        }
        score *= scale_factor;
        
        // Apply attention mask
        if (attention_mask != nullptr) {
            int mask_idx = batch_idx * seq_len * seq_len + seq_idx * seq_len + k_seq;
            score += attention_mask[mask_idx];
        }
        
        shared_scores[k_seq] = score;
        max_score = fmaxf(max_score, score);
    }
    
    // Compute softmax
    float sum_exp = 0.0f;
    for (int k_seq = 0; k_seq < seq_len; k_seq++) {
        shared_scores[k_seq] = expf(shared_scores[k_seq] - max_score);
        sum_exp += shared_scores[k_seq];
    }
    
    for (int k_seq = 0; k_seq < seq_len; k_seq++) {
        shared_scores[k_seq] /= sum_exp;
        if (attention_weights != nullptr) {
            int attn_idx = batch_idx * num_heads * seq_len * seq_len + 
                          head_idx * seq_len * seq_len + 
                          seq_idx * seq_len + k_seq;
            attention_weights[attn_idx] = shared_scores[k_seq];
        }
    }
    __syncthreads();
    
    // Compute output
    for (int d = 0; d < head_dim; d++) {
        float output_val = 0.0f;
        for (int k_seq = 0; k_seq < seq_len; k_seq++) {
            int v_idx = k_seq * head_dim + d;
            output_val += shared_scores[k_seq] * shared_value[v_idx];
        }
        
        int out_idx = batch_idx * num_heads * seq_len * head_dim + 
                      head_idx * seq_len * head_dim + 
                      seq_idx * head_dim + d;
        output[out_idx] = output_val;
    }
}

// Grouped Query Attention for Qwen2.5 (8 KV heads, 64 Q heads)
__global__ void grouped_query_attention_kernel(
    const float* query, const float* key, const float* value,
    float* output, int batch_size, int seq_len, 
    int num_q_heads, int num_kv_heads, int head_dim,
    float scale_factor) {
    
    int batch_idx = blockIdx.x;
    int q_head_idx = blockIdx.y;
    int seq_idx = threadIdx.x;
    
    if (batch_idx >= batch_size || q_head_idx >= num_q_heads || seq_idx >= seq_len) return;
    
    int kv_head_idx = q_head_idx / (num_q_heads / num_kv_heads);
    
    extern __shared__ float gqa_shared[];
    float* shared_key = gqa_shared;
    float* shared_value = gqa_shared + seq_len * head_dim;
    
    // Load corresponding KV head into shared memory
    for (int i = threadIdx.x; i < seq_len * head_dim; i += blockDim.x) {
        int k_idx = batch_idx * num_kv_heads * seq_len * head_dim + 
                    kv_head_idx * seq_len * head_dim + i;
        shared_key[i] = key[k_idx];
        shared_value[i] = value[k_idx];
    }
    __syncthreads();
    
    // Compute attention for this query head
    float max_score = -INFINITY;
    float scores[512]; // Assuming max seq_len of 512 for shared memory
    
    for (int k_seq = 0; k_seq < seq_len; k_seq++) {
        float score = 0.0f;
        for (int d = 0; d < head_dim; d++) {
            int q_idx = batch_idx * num_q_heads * seq_len * head_dim + 
                        q_head_idx * seq_len * head_dim + 
                        seq_idx * head_dim + d;
            score += query[q_idx] * shared_key[k_seq * head_dim + d];
        }
        score *= scale_factor;
        scores[k_seq] = score;
        max_score = fmaxf(max_score, score);
    }
    
    // Softmax
    float sum_exp = 0.0f;
    for (int k_seq = 0; k_seq < seq_len; k_seq++) {
        scores[k_seq] = expf(scores[k_seq] - max_score);
        sum_exp += scores[k_seq];
    }
    
    // Normalize and compute output
    for (int d = 0; d < head_dim; d++) {
        float output_val = 0.0f;
        for (int k_seq = 0; k_seq < seq_len; k_seq++) {
            float normalized_score = scores[k_seq] / sum_exp;
            output_val += normalized_score * shared_value[k_seq * head_dim + d];
        }
        
        int out_idx = batch_idx * num_q_heads * seq_len * head_dim + 
                      q_head_idx * seq_len * head_dim + 
                      seq_idx * head_dim + d;
        output[out_idx] = output_val;
    }
}

// C++ interface functions
extern "C" {
    void launch_rope_kernel(
        float* query, float* key,
        const float* cos_cache, const float* sin_cache,
        int batch_size, int seq_len, int num_heads, int head_dim,
        int max_position_embeddings, hipStream_t stream) {
        
        dim3 grid(batch_size, num_heads, seq_len);
        dim3 block(head_dim / 2);
        
        rope_kernel<<<grid, block, 0, stream>>>(
            query, key, cos_cache, sin_cache,
            batch_size, seq_len, num_heads, head_dim,
            max_position_embeddings);
    }
    
    void launch_grouped_query_attention(
        const float* query, const float* key, const float* value,
        float* output, int batch_size, int seq_len,
        int num_q_heads, int num_kv_heads, int head_dim,
        float scale_factor, hipStream_t stream) {
        
        dim3 grid(batch_size, num_q_heads);
        dim3 block(min(seq_len, MAX_THREADS_PER_BLOCK));
        
        size_t shared_mem_size = 2 * seq_len * head_dim * sizeof(float);
        
        grouped_query_attention_kernel<<<grid, block, shared_mem_size, stream>>>(
            query, key, value, output,
            batch_size, seq_len, num_q_heads, num_kv_heads, head_dim,
            scale_factor);
    }
}
